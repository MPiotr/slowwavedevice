#include "hip/hip_runtime.h"
#include "cu_mult.h"
#include "twt_0d.h"

#define PAR ParamsM
#ifndef dm_Pi
__device__ const double dm_Pi = 3.141592653589793;
#endif

static __device__ void biAverage(double *A, double *B, int p0, int datasize, int logsize)
{
	int stride = datasize;
	for (int q = 1; q <= logsize; q++)
	{
		stride = stride >> 1;
		if (p0 < stride)
		{
			A[p0] += A[p0 + stride];

		}
		else
		{
			if (p0 < 2 * stride)
			{

				B[p0 - stride] += B[p0];

			}
		}
		__syncthreads();
	}


}

inline static __device__ double cyclotronRadius(double Rcycl, double r0, double z, double hc, double phase)
{
	return Rcycl*sin(hc*z + phase) + r0;
}

__global__ void particleStep(PAR *par, double *Ar, double *Ai, int i, int k, double K, double coupling = 1, double Rshift = 0)
{
	unsigned int p0 = threadIdx.x;			unsigned int Np = blockDim.x;    //x coordinates of beam x-section, ph - cyclotron phase spread
	unsigned int x0 = threadIdx.y;			unsigned int Nx = blockDim.y;    //p -- wave phase
	unsigned int ph0 = threadIdx.z;		    unsigned int Nph = blockDim.z;

	unsigned int P = blockIdx.x;		    unsigned int NP_ = gridDim.x;  //P-grid index is for v_trans spread
	unsigned int X = blockIdx.y;			unsigned int NX = gridDim.y;
	unsigned int PC = blockIdx.z;			unsigned int NPC = gridDim.z;

	//	unsigned int p = Np*P + p0;	  	    unsigned int Np_max = Np*NP_;
	unsigned int x = Nx*X + x0;		    unsigned int Nx_max = Nx*NX;
	unsigned int ph = Nph*PC + ph0;	//	unsigned int Nph_max = Nph*NPC; 

	//index ph warps all over spreads, for now: cyclotron phase, transversal velocity.
	unsigned int NcyclPhase = Nph*NPC;
	unsigned int NvTrans = NP_;
	unsigned int ph_cyclPhase = ph;
	unsigned int ph_v_trans = P;

	unsigned int warpsize = Nx*Nph*Np;
	unsigned int log2warpsize = round(log2((double)warpsize));
	int xi = (p0 + Np*x0 + ph0*Np*Nx);           //����������� ������
	int gstride = (NX*NPC*P + NX*PC + X)*warpsize;

	//...... load problem parameters ......................... 
	double h, k1, voltage, v_trans_max;
	h = par->h;
	k1 = par->k1;
	voltage = par->voltage;
	v_trans_max = par->v_trans_max;
	// cyclotron parameters
	double h_cycl = par->h_cycl; double omega_cycl = par->omega_cycl; // ������������ �������� ����� � �������
	double ph_cyl = 2 * dm_Pi / (double)NcyclPhase*ph_cyclPhase;
	double v_trans = v_trans_max / (double)NvTrans*ph_v_trans;
	double R_cycl = (omega_cycl != 0) ? v_trans * 299.8e9 / omega_cycl : 0; //R_cycl is in mm, so 2.998e10 -> 299.8e9 
	// beam parameters
	double beamThick = par->beamThickX;
	double beamWallGap = par->wall;
	double r0 = int(x)*beamThick / (double)Nx_max + beamWallGap;  //initial positions of the particles
	double decayFactor = par->g1;
	int N = par->Nz;
	double dz = par->L / (double)N;
	double en0 = sqrt(pow(1. + voltage / 511., 2) - v_trans*v_trans);
	// .... get acess to values arrays..................
	double *d_Qk = par->Qk;   double *d_Wk = par->Wk;   //k - ���������� ���� � �������
	double *Q0 = par->Q0;   double *W0 = par->W0;   //���� and �������
	double rA = par->ar0[i];  double iA = par->ai0[i];  //��������� (��������� �� z) 	

	//array for accounting particle interseption
	int *destroyed = par->ifnotdestroyed;
	int ifnotdestroyed = destroyed[gstride + xi];

	// ..... init shared memory ...........................
	__shared__ double    sh_imJ[NS*NQ*NP];
	__shared__ double    sh_reJ[NS*NQ*NP];


	double Q, Qk;
	double W, Wk;
	double PH, EN, z, sinPH, cosPH;

	// ...... init/load phase and energy ............	
	if (i != 0) //load
	{
		Q = Q0[gstride + xi];
		W = W0[gstride + xi];
	}
	else  // or init
	{
		Q = 2.*dm_Pi / double(Np)*double(p0);
		W = 0;
		Q0[gstride + xi] = Q;
		W0[gstride + xi] = W;

	}
	// ....... at the left end of the space, set HF current to zero
	if ((i == 0) && (xi == 0))
	{
		par->avEN[NX*PC + X] = 0;
		par->int_rJ3[NX*PC + X] = 0;
		par->int_iJ3[NX*PC + X] = 0;
	}
	__syncthreads();
	// .......init k coefficient of phase and energy 

	Wk = (k > 0) ? d_Wk[4 * gstride + warpsize*(k - 1) + xi] : 0;
	Qk = (k > 0) ? d_Qk[4 * gstride + warpsize*(k - 1) + xi] : 0;

	// --------- end of initialization phase ----------

	// --------- make a step -------------------------

	z = ((double)i + K)*dz;

	double r = cyclotronRadius(R_cycl, r0, z, h_cycl, ph_cyl) +  Rshift;
	if (r < 0) 
		par->ifnotdestroyed[gstride + xi] *= 0;
	double frA = *Ar*exp(-decayFactor*r);
	double fiA = *Ai*exp(-decayFactor*r);

	frA *= coupling*double(ifnotdestroyed);
	fiA *= coupling*double(ifnotdestroyed);

	PH = Q + K*Qk;
	EN = W + K*Wk + en0;

	//	if((i < 2)&&(p0 == 0)&&(q0 < 2)) printf("%i,%i,%i\t%g\t%g\t%g\t%g\n",i, k, q0, rQ1, iQ1,  iq0, iak);

	sincos(PH, &sinPH, &cosPH);


	//...........��������� �������� ���������� �����...............................
	double DQ = (1. - (k1 / h)*EN / sqrt(EN*EN - 1.));
	double reF = frA*cosPH - fiA*sinPH;
	double imF = frA*sinPH + fiA*cosPH;
	Qk = dz*DQ;
	Wk = -dz*(rA*reF - iA*imF);
	//....................................................................

	d_Qk[4 * gstride + warpsize*k + xi] = Qk;
	d_Wk[4 * gstride + warpsize*k + xi] = Wk;

	// ===========����������=======================	

	sh_imJ[xi] = -imF / double(warpsize)*double(ifnotdestroyed);
	sh_reJ[xi] = reF / double(warpsize)*double(ifnotdestroyed);

	__syncthreads(); 


	biAverage(sh_imJ, sh_reJ, xi, warpsize, log2warpsize); // averaged values are stored in zero indexes

	//========== ������=====================

	if (xi == 0)
	{
		par->int_iJ3[NX*NPC*P + NX*PC + X] = sh_imJ[0];
		par->int_rJ3[NX*NPC*P + NX*PC + X] = sh_reJ[0];
	}
	__syncthreads();

}

static __global__ void amplitudeStep(PAR *par, int i, int k, int prevK, double kappa)
{
	unsigned int xi = threadIdx.x;

	unsigned int warpsize = blockDim.x;
	unsigned int log2warpsize = round(log2((double)warpsize));

	double *rAk = par->rAk;
	double *iAk = par->iAk;

	double *rJ = par->int_rJ3;
	double *iJ = par->int_iJ3;
	double G = par->G;
	//	double kappa = par->lossKappa;
	double delta = par->delta;

	int N = par->Nz;
	double dz = par->L / (double)N;
	//	double rak, iak;


	__shared__ double   shJr[SHARRAYSIZE];
	__shared__ double   shJi[SHARRAYSIZE];

	shJr[xi] = rJ[xi] / warpsize; //����� �� ������ ����� ����������
	shJi[xi] = iJ[xi] / warpsize;


	//	unsigned int stride = warpsize;

	__syncthreads();
	//....... ���������� �� ����� (�� ����� ��������� �� ����. ����)................
	biAverage(shJr, shJi, xi, warpsize, log2warpsize);
	//..............................................................................

	//.............. ��������� �� ��������� ���������� ����� .........................
	if (xi == 0)
	{
		par->rJ3[0] = shJr[0];
		par->iJ3[0] = shJi[0];
		double arr = par->ar0[i];
		double aii = par->ai0[i];
		if (k > 0)
		{
			rAk[k] = dz*(G*shJr[0] - kappa*(arr + prevK*rAk[k - 1]) + delta*(aii + prevK*iAk[k - 1]));
			iAk[k] = dz*(G*shJi[0] - kappa*(aii + prevK*iAk[k - 1]) - delta*(arr + prevK*rAk[k - 1]));
		}
		else
		{
			rAk[k] = dz*(G*shJr[0] - kappa*arr + delta*aii);
			iAk[k] = dz*(G*shJi[0] - kappa*aii - delta*arr);
		}
	}
	//.....................................................

	__syncthreads();

}

static __global__ void endstep(PAR *par, int i)
{
	// make an addition to the phase, energy, and amlitude in the end of one Runge-Kutta (4,4) step.
	unsigned int p0 = threadIdx.x;			unsigned int Np = blockDim.x;
	unsigned int x0 = threadIdx.y;			unsigned int Nx = blockDim.y;
	unsigned int y0 = threadIdx.z;		    unsigned int Ny = blockDim.z;

	unsigned int P = blockIdx.x;	    unsigned int NP_ = gridDim.x;
	unsigned int X = blockIdx.y;		unsigned int NX = gridDim.y;
	unsigned int Y = blockIdx.z;		unsigned int NY = gridDim.z;

	/*	unsigned int p = Np*P + p0;		    unsigned int Np_max = Np*NP_;
	unsigned int x = Nx*X + x0;		    unsigned int Nx_max = Nx*NX;
	unsigned int y = Ny*Y + y0;			unsigned int Ny_max = Ny*NY;*/

	unsigned int warpsize = Nx*Ny*Np;
	unsigned int log2warpsize = round(log2((double)warpsize));
	int xi = (p0 + Np*x0 + y0*Np*Nx);
//	unsigned int gridsize = NX*NY*NP_;
	unsigned int XI = (NX*NY*P + NX*Y + X);

	// ............ init problem parameters ...................................
	//	unsigned int xi = threadIdx.x;	    unsigned int Np = blockDim.x;
	//	unsigned int warpsize = Np;


	// .................... get access to phase, energy and current  k-coefficients
	double *d_Qk = par->Qk;		double *Q0 = par->Q0;
	double *d_Wk = par->Wk;		double *W0 = par->W0;
	double *rAk = par->rAk;    double *iAk = par->iAk;   //k - ���������� ���������

	unsigned int stride = 4 * XI*warpsize;
	unsigned int gstride = XI*warpsize;

	// ............... calculate next phase and energy .....................

	Q0[gstride + xi] += (d_Qk[stride + xi] + 2.*d_Qk[stride + xi + warpsize] + 2.*d_Qk[stride + xi + 2 * warpsize] + d_Qk[stride + xi + 3 * warpsize]) / 6.;
	W0[gstride + xi] += (d_Wk[stride + xi] + 2.*d_Wk[stride + xi + warpsize] + 2.*d_Wk[stride + xi + 2 * warpsize] + d_Wk[stride + xi + 3 * warpsize]) / 6.;


	// ............... calculate next amplitude ............................	
	if (X + xi == 0)
	{
		par->ar0[i + 1] = par->ar0[i] + (rAk[0] + 2.*rAk[1] + 2.*rAk[2] + rAk[3]) / 6.;
		par->ai0[i + 1] = par->ai0[i] + (iAk[0] + 2.*iAk[1] + 2.*iAk[2] + iAk[3]) / 6.;
	}

}

std::complex<double>  TWT_0D::solveTWT_0d(std::complex<double>  *A, double *Ar, double *Ai, double inputAmp, double lossKappa, double delta,
	double *fieldStructureRe, double *fieldStructureIm, double *mesh, double G, double enPrint, bool printField, double *lStrRe, double *lStrIm, double *qStr)
{
	int GX = Nq / NQ; int GY = Ns / NS; int GP = Nv;

	int warpsize = NP*NQ*NS;
	int gridsize = GP*GX*GY;

	double K[4] = { 0, 0.5, 0.5, 1. };

	dim3 motionwarp = dim3(NP, NQ, NS);
	dim3 motiongrid = dim3(GP, GX, GY);

	if (firstRun) {
		printf("warp setup:\n %i (wave phase) x %i (space spread) x  %i other (cycl. phase) = %i\n", NP, NQ, NS, warpsize);
		printf("grid setup:\n %i (trans. velocity) x %i (space spread) x %i (cycl. phase) = %i\n", GP, GX, GY, gridsize);
		printf("total number of particles\n, %i (wave phase), %i (space spread), %i (cyclotron spread), %i (trans_velocities)\n", NP, NQ*GX /*=Nq*/, NS*GY /*=Ns*/, GP);
		firstRun = false;
	}


	gpuErrChk(hipMemset(d_Qk, 0, 4 * sizeof(double)*Np*Nq*Ns*Nv));	gpuErrChk(hipMemset(d_W0, 0, sizeof(double)*Np*Nq*Ns*Nv));
	gpuErrChk(hipMemset(d_Wk, 0, 4 * sizeof(double)*Np*Nq*Ns*Nv));	gpuErrChk(hipMemset(d_rAk, 0, sizeof(double) * 4 * gridsize));
	gpuErrChk(hipMemset(d_Q0, 0, sizeof(double)*Np*Nq*Ns*Nv));	    gpuErrChk(hipMemset(d_iAk, 0, sizeof(double) * 4 * gridsize));

	gpuErrChk(hipMemset(d_ai0, 0, sizeof(double)*Nmax));
	gpuErrChk(hipMemset(d_ar0, 0, sizeof(double)*Nmax));
	gpuErrChk(hipMemset(d_avEN, 0, sizeof(double)));

	gpuErrChk(hipMemcpy(d_fAr, fieldStructureRe, sizeof(double), hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_fAi, fieldStructureIm, sizeof(double), hipMemcpyHostToDevice));


	double h = 2 * Pi / period*(synch_angle / 360.);
	double La = Nperiods*period*h;
	double dz = Lmax / double(Nmax);

	//TODO make separate functions for copying twt_1d params
	ParamsM par = setPar();
	par.G = G;
	par.h = h;
	par.delta = delta;
	par.lossKappa = lossKappa;
	par.beamThickX = beamHeight;
	par.beamThickY = beamHeight;
	par.omega_cycl = omega_cycl;
	par.h_cycl = h_cycl / h;
	par.v_trans_max = v_trans_max;
	par.g1 = sqrt(h*h - k1*k1);


	//	double *energy = new double[Np*Nq*Ns*Nv];
	//	double *phase = new double[Np*Nq*Ns*Nv];

	int Nstop = ceil(La / dz);
	if (Nstop >= Nmax) Nstop = Nmax - 1;
	int nd = round(period / dz);

	double z, coupl, loss, Rshift;

	Ar[0] = inputAmp; Ai[0] = 0;

	gpuErrChk(hipMemcpy(d_ai0, Ai, sizeof(double), hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_ar0, Ar, sizeof(double), hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_par, &par, sizeof(PAR), hipMemcpyHostToDevice));


	FILE *file;
	FILE *res_enr;

	//TODO:  do something with filenames
	if (printField) {
		file = fopen("twt_debug.csv", "w");
		res_enr = fopen("enPhase.csv", "w");
	}

	if (gridsize > SHARRAYSIZE)
	{
		printf("The size of the grid is larger than the shared memory array used for averaging\n");
		//TODO get rid of this limitations : add grid averaging for amplitudeStep
		return 0;
	}

	if (clinotronStructure) generateClinotronStructure(h);

	for (int i = 0; i < Nstop; i++)
	{

		for (int k = 0; k < 4; k++)
		{
			z = double(i)*dz + K[k] * dz;
			if (lStrRe) coupl = (1. - K[k])*lStrRe[i] + K[k] * lStrRe[i + 1]; else coupl = 1;
			if (qStr) loss = lossKappa / ((1. - K[k])*qStr[i] + K[k] * qStr[i + 1]); else loss = lossKappa;
			if (clinotronStructure) Rshift = (1. - K[k])*clinotronStructure[i] + K[k] * clinotronStructure[i + 1];  else Rshift = -z*clinotronAngle;
			if (!isfinite(loss)) loss = lossKappa;
			particleStep << <motiongrid, motionwarp >> >(d_par, d_fAr, d_fAi, i, k, K[k], coupl, Rshift);
			amplitudeStep << <1, gridsize >> >(d_par, i, k, K[k], loss);

			/*		double rJ, iJ;
			hipMemcpy(&rJ, d_rJ3, sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(&iJ, d_iJ3, sizeof(double), hipMemcpyDeviceToHost);
			//	fprintf(res_enr, "%g,%g,%g\n", z, rJ, iJ);
			printf("%g,%g,%g\n", z, rJ, iJ);*/
		}
		endstep << <motiongrid, motionwarp >> >(d_par, i);

	}
	gpuErrChk(hipMemcpy(Ar, d_ar0, sizeof(double)*Nstop, hipMemcpyDeviceToHost));
	gpuErrChk(hipMemcpy(Ai, d_ai0, sizeof(double)*Nstop, hipMemcpyDeviceToHost));

	for (int i = 0; i < Nstop; i++)
	{
		z = double(i)*dz;
		A[i] = cplx(Ar[i], Ai[i]);
		if (printField)
			fprintf(file, "%g,%g,%g,%g\n", z, Ar[i], Ai[i], abs(A[i]));
	}


	if (printField){
		fclose(file);
		fclose(res_enr);
	}

	return cplx(Ar[Nstop - 1], Ai[Nstop - 1]);

}
bool TWT_0D::initSolver(int nz, double lsolver)
{
	//	TWT::initSolver(nz, lsolver);

    	gpuErrChk(hipMalloc((void**)&d_fAr, NMESHMAX*sizeof(double)))
		gpuErrChk(hipMalloc((void**)&d_fAi, NMESHMAX*sizeof(double)))
		gpuErrChk(hipMalloc((void**)&d_mesh, NMESHMAX*sizeof(double)))

		printf("</font>....... End initialization\n\n");
	return 1;
}

